#include "hip/hip_runtime.h"
#include "calculate.h"
#include "stop_watch.hh"

#include <pcl/cuda/point_cloud.h>
#include <pcl/cuda/io/host_device.h>
#include <pcl/cuda/features/normal_3d.h>
#include <pcl/cuda/sample_consensus/multi_ransac.h>
#include <pcl/cuda/sample_consensus/sac_model_plane.h>

#include <cstdio>

sac_result_t ransac_cuda(
    typename pcl::PointCloud<pcl::PointXYZRGB>::Ptr cloud,
    size_t max_times,
    float distance,
    float probability) {
    
    using namespace pcl::cuda;
    
    stop_watch clock;
    
    PointCloudAOS<Host> host_cloud;
    
    const auto size = cloud->size();
    host_cloud.points.resize(size);
    host_cloud.width  = cloud->width;
    host_cloud.height = cloud->height;
    for (size_t i = 0; i < size; ++i) {
        auto        p0 = cloud->points[i];
        PointXYZRGB p;
        p.x            = p0.x;
        p.y            = p0.y;
        p.z            = p0.z;
        host_cloud.points[i] = p;
    }
    
    using sac_model_t = SampleConsensusModelPlane<Device>;
    using sac_t = MultiRandomSampleConsensus<Device>;
    
    auto data = toStorage<Host, Device>(host_cloud);
    data->height = cloud->height;
    data->width  = cloud->width;
    
    typename sac_model_t::Ptr sac_model(new sac_model_t(data));
    
//    auto normals = computeFastPointNormals<Device>(data);
//    sac_model->setNormals(normals);
    
    sac_t sac(sac_model, distance);
    sac.setMinimumCoverage(0.9);
    sac.setMaximumBatches(1);
    sac.setIerationsPerBatch(2000);
    //    sac.setMaxIterations(max_times);
    //    sac.setProbability(probability);
    
    auto calculate_time = clock.seconds();
    auto success        = sac.computeModel(0);
    auto time           = clock.seconds();
    
    if (!success) return {-1, -1};
    
    sac_result_t result{time - calculate_time, time};
    
    auto temp = sac.getAllModelCoefficients();
    
    if (!temp.empty()) {
        result.model[0] = temp[0].x;
        result.model[1] = temp[0].y;
        result.model[2] = temp[0].z;
        result.model[3] = temp[0].w;
    }
    
    //    Device<float>::type device_vector(4);
    //    sac.getModelCoefficients(device_vector);
    //    Host<float>::type host_vector = device_vector;
    //    thrust::copy(device_vector.begin(), device_vector.end(), host_vector.begin());
    //    result.rate = static_cast<double>(sac_model->countWithinDistance(device_vector, distance)) / cloud->size();
    
    return result;
}
