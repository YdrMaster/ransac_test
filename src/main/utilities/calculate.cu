#include "hip/hip_runtime.h"
#include "calculate.h"
#include "stop_watch.hh"

#include <pcl/cuda/point_cloud.h>
#include <pcl/cuda/io/host_device.h>
#include <pcl/cuda/sample_consensus/ransac.h>
#include <pcl/cuda/sample_consensus/sac_model_plane.h>

sac_result_t ransac_cuda(
    typename pcl::PointCloud<pcl::PointXYZRGB>::Ptr cloud,
    size_t max_times,
    float distance,
    float probability) {
    
    using namespace pcl::cuda;
    
    stop_watch clock;
    
    PointCloudAOS<Host> host_cloud;
    
    const auto size = cloud->size();
    host_cloud.points.resize(size);
    host_cloud.width  = cloud->width;
    host_cloud.height = cloud->height;
    for (size_t i = 0; i < size; ++i) {
        auto        p0 = cloud->points[i];
        PointXYZRGB p;
        p.x            = p0.x;
        p.y            = p0.y;
        p.z            = p0.z;
        host_cloud.points[i] = p;
    }
    
    auto data = toStorage<Host, Device>(host_cloud);
    using sac_model_t = SampleConsensusModelPlane<Device>;
    using sac_t = RandomSampleConsensus<Device>;
    typename sac_model_t::Ptr sac_model(new sac_model_t(data));
    sac_t                     sac(sac_model, distance);
    sac.setMaxIterations(max_times);
    sac.setProbability(probability);
    
    auto calculate_time = clock.seconds();
    auto success        = sac.computeModel();
    auto time           = clock.seconds();
    
    if (!success) return {};
    
    sac_result_t result{
        time - calculate_time,
        time
    };
    
    Device<float>::type device_vector(4);
    Host<float>::type   host_vector(4);
    sac.getModelCoefficients(device_vector);
    thrust::copy(device_vector.begin(), device_vector.end(), host_vector.begin());
    //    result.rate = static_cast<double>(sac_model->countWithinDistance(device_vector, distance)) / cloud->size();
    
//    for (int j = 0; j < 4; ++j) result.model[j] = host_vector[j];
    
    return result;
}
