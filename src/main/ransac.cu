#include "hip/hip_runtime.h"
﻿//
// Created by User on 2019/6/4.
//

#ifndef RANSAC_RANSAC_CU
#define RANSAC_RANSAC_CU


#include "hip/hip_runtime.h"
#include ""

#include <array>
#include <algorithm>
#include "models/model_t.hpp"
#include "random_engine.hpp"

template<class _model_t>
struct ransac_result_t {
    _model_t            model{};
    std::vector<size_t> inliers{};
    float               rate = 0;
};

template<class ptr_i_t, class ptr_o_t, class function_t>
__global__ void map(ptr_i_t ptr_i, ptr_o_t ptr_o, function_t function) {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    ptr_o[i] = function(ptr_i[i]);
}

template<class _model_t>
ransac_result_t<_model_t>
ransac(const std::vector<typename _model_t::super_t::_point_t> &data,
       float threshold,
       float success_rate = 0.99f,
       size_t max_times = std::numeric_limits<size_t>::max(),
       const _model_t &guess = _model_t{}
) {
    using tp = typename _model_t::super_t::_point_t;
    using ti = decltype(data.size());
    
    const auto size         = data.size(),
               success_size = static_cast<ti>(success_rate * size);
    
    if (size < 2 * _model_t::super_t::size_to_make)
        throw std::logic_error("samples too little");
    
    // 随机数引擎
    random_engine<ti>
         random(0, size - 1);
    // 模型初始化器
    std::array<tp, _model_t::super_t::size_to_make>
         initialize_list{};
    // 缓存
    tp   *point_buffer = nullptr;
    bool *check_buffer = nullptr;
    
    hipMallocManaged(&point_buffer, size * sizeof(tp));
    hipMallocManaged(&check_buffer, size * sizeof(bool));
		
    ti       count      = 0;
    _model_t best_model = guess,
             model{};
    
    if (best_model.is_valid()) {
        map<<<(size - 1) / 32 + 1, 32>>>(point_buffer, check_buffer,
                                         [=](const tp &point) { return std::abs(best_model(point)) < threshold; });
	hipDeviceSynchronize();
    
        count = std::count(check_buffer, check_buffer + size, true);
    }
    
    for (; max_times > 0 && count < success_size; --max_times) {
        for (ti i = 0; i < initialize_list.size(); ++i)
            initialize_list[i] = data[random()];
        
        model.make(initialize_list);
        if (!model.is_valid() || (best_model.is_valid() && model == best_model))
            continue;
        
        map<<<(size - 1) / 32 + 1, 32>>>(point_buffer, check_buffer,
                                         [=](const tp &point) { return std::abs(best_model(point)) < threshold; });
	hipDeviceSynchronize();
        
        ti temp = std::count(check_buffer, check_buffer + size, true);
        if (temp > count) {
            count      = temp;
            best_model = model;
        }
    }
    
    hipFree(point_buffer);
    hipFree(check_buffer);

    // 局内点
    std::vector<ti> inliers(count);
    
    auto    ptr = inliers.begin();
    for (ti i   = 0; i < size; ++i)
        if (check_buffer[i]) *ptr++ = i;
    
    return {best_model, inliers, static_cast<float>(inliers.size()) / size};
}


#endif // RANSAC_RANSAC_CU
